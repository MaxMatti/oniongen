#include "hip/hip_runtime.h"
#include <iostream>

#include "helpers.hh"

#define CUDA_CHECK(cmd) cuda_check(cmd, __FILE__, __LINE__, 0)
#define CUDA_CHECK_FATAL(cmd) cuda_check(cmd, __FILE__, __LINE__, 1)
#define CUDA_CHECK_KERNEL(fatal) cuda_check(hipGetLastError(), __FILE__, __LINE__, fatal)

void cuda_check(hipError_t error, const char* file, int line, int fatal) {
	if (error != hipSuccess) {
		std::cerr << "Caught Cuda Error: " << hipGetErrorString(error) << " at " << file << ":" << line << std::endl;
		if (fatal) {
			std::cerr << "Error marked as fatal, exiting." << std::endl;
			exit(-1);
		}
	}
}

namespace gpu {
	// Swaps the endian of uint32_t variables. CUDA 7.5 doesn't like templates. Swapped "T" for "uint32_t" from http://stackoverflow.com/a/4956493
	__device__ uint32_t swap_endian(uint32_t u) {
		static_assert (CHAR_BIT == 8, "CHAR_BIT != 8");
		union {
			uint32_t u;
			unsigned char u8[sizeof(uint32_t)];
		} source, dest;
		source.u = u;
		for (unsigned int k = 0; k < sizeof(uint32_t); k++) {
			dest.u8[k] = source.u8[sizeof(uint32_t) - k - 1];
		}
		return dest.u;
	}

	// Swaps the endian of uint64_t variables. CUDA 7.5 doesn't like templates. Swapped "T" for "uint64_t" from http://stackoverflow.com/a/4956493
	__device__ uint64_t swap_endian(uint64_t u) {
		static_assert (CHAR_BIT == 8, "CHAR_BIT != 8");
		union {
			uint64_t u;
			unsigned char u8[sizeof(uint64_t)];
		} source, dest;
		source.u = u;
		for (unsigned int k = 0; k < sizeof(uint64_t); k++) {
			dest.u8[k] = source.u8[sizeof(uint64_t) - k - 1];
		}
		return dest.u;
	}

	template <typename T> __host__ __device__ T min(T a, T b) {
		return a > b ? b : a;
	}

	// implementation of the s function as described in section 3. of spec
	__device__ std::uint32_t sha1_helper_s(std::uint32_t input, unsigned char offset) {
		return (input << offset) | (input >> (32 - offset));
	}

	// implementation of the f function as described in section 5. of spec
	__device__ std::uint32_t sha1_helper_f(unsigned char nr, const std::uint32_t& b, const std::uint32_t& c, const std::uint32_t& d) {
		if (nr < 20) {
			return (b & c) | ((b ^ 0xFFFFFFFF) & d);
		} else if (nr < 40) {
			return b ^ c ^ d;
		} else if (nr < 60) {
			return (b & c) | (b & d) | (c & d);
		} else if (nr < 80) {
			return b ^ c ^ d;
		} else {
			return 0;
		}
	}

	// representation of the K variables as described in section 5. of spec
	__device__ std::uint32_t sha1_helper_K(unsigned char nr) {
		if (nr < 20) {
			return 0x5A827999;
		} else if (nr < 40) {
			return 0x6ED9EBA1;
		} else if (nr < 60) {
			return 0x8F1BBCDC;
		} else if (nr < 80) {
			return 0xCA62C1D6;
		} else {
			return 0;
		}
	}

	// calculates the sha1 sum
	__global__ void d_sha1(unsigned char* input_buffer, unsigned int input_buffer_size, unsigned char* output, unsigned int threads) {
		unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
		if (x >= threads) {
			return;
		}
		output += x * 20;
		// in case the machine uses big endian we need to swap some bytes later:
		bool convert_endians;
		{
			std::uint32_t endian = 0x08040201;
			if (*reinterpret_cast<unsigned char*>(&endian) == 1) {
				convert_endians = 1;
			} else if (*reinterpret_cast<unsigned char*>(&endian) == 8) {
				convert_endians = 0;
			} else {
				return;
			}
		}
		// copying input_buffer to own storage area with larger size:
		const unsigned int input_size = (input_buffer_size + 72) & 0xFFFFFFC0;
		// 72 because 512bit blocks (64bytes) and ending in length (64bit aka 8 bytes) and 1 byte because of padding starting with 0b10000000
		// 72 = 63 + 8 + 1 (63 instead of 64 because we're counting from 0)

		unsigned char* input = input_buffer + x * input_size;
		if (((int) input_size) - input_buffer_size > 64) {
			memset(input_buffer + threads * (input_size - 128) + (128 + input_buffer_size - input_size) + (x * 64) + 1, 0, (((int) input_size) - input_buffer_size - 64) * sizeof(char));
			memset(input_buffer + threads * (input_size - 64) + (x * 64), 0, 59 * sizeof(char));
		} else {
			memset(input_buffer + threads * (input_size - 64) + (64 + input_buffer_size - input_size) + (x * 64) + 1, 0, (input_size - input_buffer_size - 5) * sizeof(char));
		}

		// 4. filling up input buffer according to spec
		*(input_buffer + threads * (input_size - 64) + (64 + input_buffer_size - input_size) + (x * 64)) = 0x80; // set first bit to 1, others to 0

		// start scope because we don't need those variables later:
		{
			std::uint64_t tmp;
			if (convert_endians) { // convert endianness in case of big endian
				tmp = swap_endian(((std::uint64_t) input_buffer_size) << 3);
			} else {
				tmp = input_buffer_size << 3;
			}
			memcpy(input + input_size - 8, &tmp, 8);
		}

		// 6.1 actual hash algorithm:

		// initializing result buffer (h0-h4):
		// std::uint32_t result[5];
		std::uint32_t* result = reinterpret_cast<std::uint32_t*>(output);
		result[0] = 0x67452301;
		result[1] = 0xefcdab89;
		result[2] = 0x98badcfe;
		result[3] = 0x10325476;
		result[4] = 0xc3d2e1f0;

		// initializing block buffer, tmp "word" and "words" A-E as described in 6.2
		std::uint32_t current_block[80];
		std::uint32_t tmp[6] = {0, 0, 0, 0, 0, 0}; // tmp and then a-e

		// processing block by block
		for (unsigned int i = 0; i < min(input_size, (unsigned int) 0); i += 64) {

			// copy current block to buffer
			memcpy(current_block, input_buffer + i * threads + x * 64, 64 * sizeof(unsigned char));

			// convert endianness in case of big endian
			for (unsigned int j = 0; j < 64 && convert_endians; ++j) {
				current_block[j] = swap_endian(current_block[j]);
			}

			// 6.2 (b) calculate the rest of the current block
			for (unsigned int j = 16; j < 80; ++j) {
				current_block[j] = sha1_helper_s(current_block[j - 3] ^ current_block[j - 8] ^ current_block[j - 14] ^ current_block[j - 16], 1);
			}

			// 6.2 (c) fill a-e
			memcpy(tmp + 1, result, 5 * sizeof(int32_t));

			// 6.2 (d) wobble around
			for (unsigned char j = 0; j < 80; ++j) {
				tmp[0] = sha1_helper_s(tmp[1], 5) + sha1_helper_f(j, tmp[2], tmp[3], tmp[4]) + tmp[5] + current_block[j] + sha1_helper_K(j);
				tmp[5] = tmp[4];
				tmp[4] = tmp[3];
				tmp[3] = sha1_helper_s(tmp[2], 30);
				tmp[2] = tmp[1];
				tmp[1] = tmp[0];
			}

			// 6.2 (e) write output of wobbling on top of current result
			result[0] += tmp[1];
			result[1] += tmp[2];
			result[2] += tmp[3];
			result[3] += tmp[4];
			result[4] += tmp[5];
		}
		// convert endianness in case of big endian
		for (unsigned int j = 0; j < 5 && convert_endians; ++j) {
			result[j] = swap_endian(result[j]);
		}
	}

	void sha1_allocate(unsigned int device, unsigned int total_input_size, unsigned int total_output_size, unsigned char** d_input_buffer, unsigned char** d_output_buffer) {
		CUDA_CHECK(hipSetDevice(device));
		CUDA_CHECK_FATAL(hipMalloc(d_input_buffer, total_input_size));
		CUDA_CHECK_FATAL(hipMalloc(d_output_buffer, total_output_size));
	}

	void sha1_prepare(unsigned char* h_input_buffer, unsigned char* d_input_buffer, unsigned int input_buffer_size, unsigned char* h_output, unsigned char* d_output, unsigned int threads) {
		const unsigned int input_size = (input_buffer_size + 72) & 0xFFFFFFC0;

		CUDA_CHECK_FATAL(hipMemcpy(d_input_buffer, h_input_buffer, input_size * threads * sizeof(char), hipMemcpyHostToDevice));
	}

	void sha1(unsigned char* input_buffer, unsigned int input_buffer_size, unsigned char* output, unsigned int threads) {
		unsigned int blocksize = 256;
		dim3 dimBlock(blocksize);
		dim3 dimGrid(helpers::fastCeil(threads, blocksize));

		d_sha1<<<dimGrid, dimBlock>>>(input_buffer, input_buffer_size, output, threads);
		CUDA_CHECK_KERNEL(1);
	}

	void sha1_cleanup(unsigned char* h_input_buffer, unsigned char* d_input_buffer, unsigned int input_buffer_size, unsigned char* h_output, unsigned char* d_output, unsigned int threads) {
		CUDA_CHECK_FATAL(hipMemcpy(h_output, d_output, 20 * threads * sizeof(char), hipMemcpyDeviceToHost));
	}

	void sha1_free(unsigned int total_input_size, unsigned int total_output_size, unsigned char** d_input_buffer, unsigned char** d_output_buffer) {
		CUDA_CHECK(hipFree(*d_input_buffer));
		CUDA_CHECK(hipFree(*d_output_buffer));
		CUDA_CHECK(hipDeviceReset());
	}
}
